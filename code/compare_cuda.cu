#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include "cuhe/CuHE.h"
using namespace cuHE;
#include <iostream>
#include <time.h>
#include <random>


__global__ void sayHi(){
    printf("==kernel code== Hi there, this is Ardhi\n");
}



int compare_cuda(NTL::ZZX& c1, NTL::ZZX& c2) {
cout<<"inside compare cuda\n";

#if 1
ZZX a,b, c; //NTL Lib

//set polynomial a to x^2 + 3x + 1
SetCoeff(a, 0, 1); //1
SetCoeff(a, 1, 3); //3x + 1
SetCoeff(a, 2, 1); //x^2 + 3x + 1

//set polynomial a to 2x^2 + 2x + 2
SetCoeff(b, 0, 2); //2
SetCoeff(b, 1, 2); //2x + 2
SetCoeff(b, 2, 2); //2x^2 + 2x + 2

//c = a+b = 3x^2 + 5x + 3
c = a+b;

//cuHE initilization with DHS
// setParameters(d, p, w, min, cut, m);
setParameters(5, 2, 1, 61, 20, 8191);

cout<<"Poly a: "<<a<<endl;
cout<<"Poly b: "<<b<<endl;
cout<<"Poly c: "<<c<<endl;

multiGPUs(1);

CuPtxt* cu_a = new CuPtxt;
CuPtxt* cu_b = new CuPtxt;
CuPtxt* cu_c = new CuPtxt;

//convert ZZX to CuPtxt representation
cu_a->zRep(a);
cu_b->zRep(b);
cu_c->zRep(c);


cu_a->x2n();
cu_b->x2n();
cu_c->x2n();
#endif

cXor(*cu_c, *cu_a, *cu_b); // use cXor

// cu_c->x2z();
// ZZX result = cu_c->zRep();

// cout<<"result : "<<result<<endl;

sayHi<<<3, 3>>> ();

	return 0;
}
